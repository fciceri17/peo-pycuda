
#include <hip/hip_runtime.h>
// Getter of the D set
// The D set is defined as the nodes in CuB that have more than 3/5*|C| neighbors in C
__global__ void stratify_lowdegree_getD(float *CuB, float *C, int *indptr, int *indices, int n, float c, float *D)
{
    const int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(i >= n) return;
    D[i] = 0;
    if(CuB[i] == 0) return;

    int d = 0;
    for(int j = indptr[i]; j < indptr[i+1]; j++){
        if(C[indices[j]]){
            d += 1;
        }
    }

    if(d >= 3 / 5 * c){
        D[i] = 1;
    }
}