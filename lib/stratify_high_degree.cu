
#include <hip/hip_runtime.h>

__global__ void compute_adjacent_nodes(int *indptr, int *indices, float *in_component, float *update_values, float *adjancencies, int n)
{
    const int i = threadIdx.x;
    if(update_values[i]==0)
        return;
    int offset = i*n;
    for(int j = indptr[i]; j < indptr[i+1]; j++)
        if(in_component[indices[j]] == 1)
            adjancencies[offset+indices[j]] = 1;

}

__global__ void logic_and(float *arr_a, float *arr_b, float *arr_dest)
{
    const int i = threadIdx.x;
    arr_dest[i] = arr_a[i] && arr_b[i];

}