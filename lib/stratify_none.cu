
#include <hip/hip_runtime.h>
// Getter of the D set
// The D set is defined as the nodes in C that have more than 3/5*|C| neighbors in C
__global__ void stratify_none_getD(float *C, int *indptr, int *indices, int n, float c, float *D)
{
    const int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(i >= n) return;
    D[i] = 0;
    if(C[i] == 0) return;

    int d = 0;
    for(int j = indptr[i]; j < indptr[i+1]; j++){
        if(C[indices[j]]){
            d += 1;
        }
    }

    if(d > c * 3 / 5 ){
        D[i] = 1;
    }
}