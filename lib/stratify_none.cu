
#include <hip/hip_runtime.h>

__global__ void stratify_none_getD(float *is_class_component, int *indptr, int *indices, int n, float c, float *D)
{
    const int i = threadIdx.x;
    D[i] = 0;
    if(is_class_component[i] == 0) return;

    int d = 0;
    for(int j = indptr[i]; j < indptr[i+1]; j++){
        if(is_class_component[indices[j]]){
            d += 1;
        }
    }

    if(d >= 3 / 5 * c){
        D[i] = 1;
    }
}

__global__ void stratify_none_getC_D(float *is_class_component, int *indptr, int *indices, int n, float *D, float *C_D)
{
    const int i = threadIdx.x;
    C_D[i] = 0;

    if(is_class_component[i] == 0) return;

    if(D[i] == 0) C_D[i] = 1;
}