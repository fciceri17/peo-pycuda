
#include <hip/hip_runtime.h>
// Getter of the D set
// The D set is defined as the nodes in C that have more than 3/5*|C| neighbors in C
__global__ void stratify_none_getD(float *C, int *indptr, int *indices, int n, float c, float *D)
{
    const int i = threadIdx.x;
    D[i] = 0;
    if(C[i] == 0) return;

    int d = 0;
    for(int j = indptr[i]; j < indptr[i+1]; j++){
        if(C[indices[j]]){
            d += 1;
        }
    }

    if(d >= 3 / 5 * c){
        D[i] = 1;
    }
}