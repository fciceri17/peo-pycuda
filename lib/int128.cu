
#include <hip/hip_runtime.h>
typedef struct {

  unsigned long long int lo;

  unsigned long long int hi;

} my_uint128;

__host__ __device__ inline bool operator==(const my_uint128& lhs, const my_uint128& rhs){ return lhs.lo == rhs.lo && lhs.hi == rhs.hi; }
__host__ __device__ inline bool operator>(const my_uint128& lhs, const my_uint128& rhs){ return (lhs.lo > rhs.lo && lhs.hi == rhs.hi) || lhs.hi > rhs.hi; }

__host__ __device__ my_uint128 int_to_my_uint128(int a)
{
    my_uint128 res;
    res.lo = a;
    res.hi = 0;
    return res;
}

__host__ __device__ my_uint128 llint_to_uint128(unsigned long long int a, unsigned long long int b)
{
    my_uint128 res;
    res.hi = a;
    res.lo = b;
    return res;
}


__host__ __device__ my_uint128 add_my_uint128 (my_uint128 a, my_uint128 b)

{

  my_uint128 res;

  res.lo = a.lo + b.lo;

  res.hi = a.hi + b.hi + (res.lo < a.lo);

  return res;

}



__host__ __device__ my_uint128 sub_my_uint128 (my_uint128 a, my_uint128 b)

{

  my_uint128 res;

  res.lo = a.lo - b.lo;

  res.hi = a.hi - b.hi - (res.lo > a.lo);

  return res;

}



__host__ __device__ my_uint128 shl_my_uint128 (my_uint128 a, int s)

{

  if (s) {

    a.hi = (a.hi << s) | (a.lo >> (64 - s));

    a.lo =  a.lo << s;

  }

  return a;

}



__host__ __device__ my_uint128 mul10_my_uint128 (my_uint128 a)

{

  my_uint128 s, t;

  s = shl_my_uint128 (a, 3);

  t = shl_my_uint128 (a, 1);

  return add_my_uint128 (s, t);

}



static const my_uint128 pwrten [] =

{

  {0x0000000000000001, 0x0000000000000000}, /* 10**0  */

  {0x000000000000000a, 0x0000000000000000}, /* 10**1  */

  {0x0000000000000064, 0x0000000000000000}, /* 10**2  */

  {0x00000000000003e8, 0x0000000000000000}, /* 10**3  */

  {0x0000000000002710, 0x0000000000000000}, /* 10**4  */

  {0x00000000000186a0, 0x0000000000000000}, /* 10**5  */

  {0x00000000000f4240, 0x0000000000000000}, /* 10**6  */

  {0x0000000000989680, 0x0000000000000000}, /* 10**7  */

  {0x0000000005f5e100, 0x0000000000000000}, /* 10**8  */

  {0x000000003b9aca00, 0x0000000000000000}, /* 10**9  */

  {0x00000002540be400, 0x0000000000000000}, /* 10**10 */

  {0x000000174876e800, 0x0000000000000000}, /* 10**11 */

  {0x000000e8d4a51000, 0x0000000000000000}, /* 10**12 */

  {0x000009184e72a000, 0x0000000000000000}, /* 10**13 */

  {0x00005af3107a4000, 0x0000000000000000}, /* 10**14 */

  {0x00038d7ea4c68000, 0x0000000000000000}, /* 10**15 */

  {0x002386f26fc10000, 0x0000000000000000}, /* 10**16 */

  {0x016345785d8a0000, 0x0000000000000000}, /* 10**17 */

  {0x0de0b6b3a7640000, 0x0000000000000000}, /* 10**18 */

  {0x8ac7230489e80000, 0x0000000000000000}, /* 10**19 */

  {0x6bc75e2d63100000, 0x0000000000000005}, /* 10**20 */

  {0x35c9adc5dea00000, 0x0000000000000036}, /* 10**21 */

  {0x19e0c9bab2400000, 0x000000000000021e}, /* 10**22 */

  {0x02c7e14af6800000, 0x000000000000152d}, /* 10**23 */

  {0x1bcecceda1000000, 0x000000000000d3c2}, /* 10**24 */

  {0x161401484a000000, 0x0000000000084595}, /* 10**25 */

  {0xdcc80cd2e4000000, 0x000000000052b7d2}, /* 10**26 */

  {0x9fd0803ce8000000, 0x00000000033b2e3c}, /* 10**27 */

  {0x3e25026110000000, 0x00000000204fce5e}, /* 10**28 */

  {0x6d7217caa0000000, 0x00000001431e0fae}, /* 10**29 */

  {0x4674edea40000000, 0x0000000c9f2c9cd0}, /* 10**30 */

  {0xc0914b2680000000, 0x0000007e37be2022}, /* 10**31 */

  {0x85acef8100000000, 0x000004ee2d6d415b}, /* 10**32 */

  {0x38c15b0a00000000, 0x0000314dc6448d93}, /* 10**33 */

  {0x378d8e6400000000, 0x0001ed09bead87c0}, /* 10**34 */

  {0x2b878fe800000000, 0x0013426172c74d82}, /* 10**35 */

  {0xb34b9f1000000000, 0x00c097ce7bc90715}, /* 10**36 */

  {0x00f436a000000000, 0x0785ee10d5da46d9}, /* 10**37 */

};

#define MAX_PWR ((int)(sizeof(pwrten)/sizeof(pwrten[0]))-1)

#define DIGITS  (MAX_PWR+1)



void cvt_my_uint128_to_str (my_uint128 a, char *cp) 

{

  my_uint128 t;

  int pwr, bit, non_zero, digit, remainder_neg;

  non_zero = 0;

  for (pwr = MAX_PWR; pwr >= 0; pwr--) {

    digit = 0;

    for (bit = 3; bit >= 0; bit--) {

      t = shl_my_uint128 (pwrten[pwr], bit);

      a = sub_my_uint128 (a, t);

      remainder_neg = ((long long int)a.hi) < 0;

      digit = (digit << 1) | !remainder_neg;

      if (remainder_neg) {

        a = add_my_uint128 (a, t);

      }

    }

    non_zero |= digit;

    if (non_zero || pwr == 0) {

      *cp++ = '0' + digit;

    }

    *cp = 0;

  }

}



my_uint128 cvt_str_to_my_uint128 (char *cp)

{

  my_uint128 a = {0, 0};

  my_uint128 t = {0, 0};

  while (*cp) {

    a = mul10_my_uint128 (a);

    t.lo = *cp++ - '0';

    a = add_my_uint128 (a, t);

  }

  return a;

}
